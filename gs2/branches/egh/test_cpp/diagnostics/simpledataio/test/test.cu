#include <cstdlib>
#include "hipfft/hipfft.h"
#include "simpledataio_cuda.h"

int main (int argc, char * argv){
	struct sdatio_file sdatfile;
	double yvar[2] = {0.1,0.3};
	int iy[2] = {1,2};
	double phivar[3][2] = {{0.1,0.3}, {2.0, 4.0}, {-1.0, 3.6}};
	double t;
	double phi_tvar[2];
	int i;

	hipDoubleComplex compvar[3];

	sdatio_debug = 0;

	sdatio_createfile(&sdatfile, "testfile.cdf");

	sdatio_add_dimension(&sdatfile, "r", 2, "Real and imaginary parts", "(none)");
	sdatio_add_dimension(&sdatfile, "x", 3, "The x coordinate", "m");
	sdatio_add_dimension(&sdatfile, "y", 2, "The y coordinate", "m");
	sdatio_add_dimension(&sdatfile, "t", SDATIO_UNLIMITED, "The time coordinate", "s");
	sdatio_print_dimensions(&sdatfile);

	sdatio_create_variable(&sdatfile, SDATIO_DOUBLE, "comp", "xr", "A complex variable", "(none)");
	sdatio_create_variable(&sdatfile, SDATIO_DOUBLE, "phi", "xy", "Some potential", "Vm");
	sdatio_create_variable(&sdatfile, SDATIO_DOUBLE, "phi_t", "ty", "Some potential as a function of y and time", "Vm");
	sdatio_create_variable(&sdatfile, SDATIO_DOUBLE, "y", "y", "Values of the y coordinate", "m");
	sdatio_create_variable(&sdatfile, SDATIO_DOUBLE, "t", "t", "Values of the time coordinate", "m");
	sdatio_create_variable(&sdatfile, SDATIO_INT, "iky", "y", "y index values", "(none)");
	sdatio_print_variables(&sdatfile);


	for (i=0;i<6;i++){
		t = 0.3 + i;
		phi_tvar[0] = 4 + i/2.0;
		phi_tvar[1] = 6 + i*3.0; 
		sdatio_write_variable(&sdatfile, "t", &t);
		sdatio_write_variable(&sdatfile, "phi_t", &phi_tvar);
		sdatio_increment_start(&sdatfile, "t");
		//if (i>2) abort();
	}

	sdatio_write_variable(&sdatfile, "y", &yvar[0]);
	sdatio_write_variable(&sdatfile, "iky", &iy[0]);
	sdatio_write_variable(&sdatfile, "phi", &phivar[0]);

	compvar[0].x = 1.0;
	compvar[0].y = -1.0;
	compvar[1].x = 2.0;
	compvar[1].y = -2.0;
	compvar[2].x = 3.0;
	compvar[2].y = -3.0;

	sdatio_write_variable(&sdatfile, "comp", &compvar[0]);


	sdatio_close(&sdatfile);

	printf("Success!\n");
	return 0;
}
